#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <vector>
#include "RNG_LCG.hu"
#include "RNG_TAUS.hu"
#include "RNG.hu"
#include "RNG_COMB.hu"

using namespace std;

#define N_PASSI 5 //N_PASSI indica quanti numeri casuali genera ogni thread

struct thread_seed{
    double a, b, c, d;

    thread_seed(){
        a = rand();
        b = rand();
        c = rand();
        d = rand();
    }
};
/*
__global__ void CombinedGenerator(thread_seed* v_thread_seed, double* dev_average_thread){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    RNG* rng_Comb = new RNG_COMB(v_thread_seed[tid].a, v_thread_seed[tid].b, v_thread_seed[tid].c, v_thread_seed[tid].d);

	double sum = 0;
    for(int i=0; i<N_PASSI; i++){
        sum += rng_Comb->get_uniform();
    }
	dev_average_thread[tid] = sum/N_PASSI;
}
*/

int main(int argc, char**argv){
    int N = atoi(argv[1]);                     //N e' il numero di scenari generato
    srand(time(0));

    thread_seed* v_thread_seed = new thread_seed[N];
    RNG* rng_Lcg1 = new RNG_LCG(v_thread_seed[0].a);
    RNG* rng_Lcg2 = new RNG_LCG(v_thread_seed[0].b);
    RNG* rng_Lcg3 = new RNG_LCG(v_thread_seed[0].c);
    RNG* rng_Lcg4 = new RNG_LCG(v_thread_seed[0].d);
  
    thread_seed* dev_v_thread_seed = new thread_seed[N];

    double average_thread[N];             //Vettore di medie per ogni thread
    double* dev_average_thread;

    for(int i=0; i<N; i++){
        v_thread_seed[i].a = rng_Lcg1->get_uniform();
        v_thread_seed[i].b = rng_Lcg2->get_uniform();
        v_thread_seed[i].c = rng_Lcg3->get_uniform();
        v_thread_seed[i].d = rng_Lcg4->get_uniform();
    }

    int THREADS_PER_BLOCK = 1024;
    int N_BLOCK = N/1024 + 1;
    
/*  SIMULAZIONE KERNEL FUNZIONA SU CPU */
	double average_thread[N];
    RNG* rng_Comb;
    for(int tid = 0; tid<N; tid++){
		rng_Comb = new RNG_COMB(v_thread_seed[tid].a, v_thread_seed[tid].b, v_thread_seed[tid].c, v_thread_seed[tid].d);

		double sum = 0;
		for(int i=0; i<N_PASSI; i++){
		    sum += rng_Comb->get_uniform();
		}
		average_thread[tid] = sum/N_PASSI;
	}

/*
    hipMalloc((void**)&dev_v_thread_seed,N*sizeof(thread_seed));
    hipMalloc((void**)&dev_average_thread,N*sizeof(double));

    hipMemcpy(dev_v_thread_seed,v_thread_seed,N*sizeof(thread_seed),hipMemcpyHostToDevice);

    CombinedGenerator<<<N_BLOCK,THREADS_PER_BLOCK>>>(dev_v_thread_seed, dev_average_thread);

    hipMemcpy(average_thread,dev_average_thread,N*sizeof(double),hipMemcpyDeviceToHost);


    for(int i=0; i<N; i++){
            cout<<average_thread[i]<<" "<<endl;
    }

	hipFree(dev_v_thread_seed);
	hipFree(dev_average_thread);
*/
}


