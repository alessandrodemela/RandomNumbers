#include "header.hu"


int main(int argc, char**argv){
    int N = atoi(argv[1]);                     //N e' il numero di scenari generato
    srand(time(0));

    thread_seed* v_thread_seed = new thread_seed[N];
    RNG* rng_Lcg = new RNG_LCG(rand());

    for(int i=0; i<N; i++){
        v_thread_seed[i].a = rng_Lcg->get_uniform();
        v_thread_seed[i].b = rng_Lcg->get_uniform();
        v_thread_seed[i].c = rng_Lcg->get_uniform();
        v_thread_seed[i].d = rng_Lcg->get_uniform();
    }

    thread_seed* dev_v_thread_seed = new thread_seed[N];

    double average_thread[N];             //Vettore di medie per ogni thread
    double* dev_average_thread;

    int THREADS_PER_BLOCK = 1024;
    int N_BLOCK = N/1024 + 1;

/*  SIMULAZIONE KERNEL FUNZIONA SU CPU
	double average_thread[N];
    RNG* rng_Comb;
    for(int tid = 0; tid<N; tid++){
		rng_Comb = new RNG_COMB(v_thread_seed[tid].a, v_thread_seed[tid].b, v_thread_seed[tid].c, v_thread_seed[tid].d);

		double sum = 0;
		for(int i=0; i<N_PASSI; i++){
		    sum += rng_Comb->get_uniform();
		}
		average_thread[tid] = sum/N_PASSI;
	}
*/

    hipMalloc((void**)&dev_v_thread_seed,N*sizeof(thread_seed));
    hipMalloc((void**)&dev_average_thread,N*sizeof(double));

    hipMemset(dev_average_thread,0,N*sizeof(double));

    hipMemcpy(dev_v_thread_seed,v_thread_seed,N*sizeof(thread_seed),hipMemcpyHostToDevice);

    int N_BLOCK = N/THREADS_PER_BLOCK + 1;
    CombinedGenerator<<<N_BLOCK,THREADS_PER_BLOCK>>>(dev_v_thread_seed, dev_average_thread);

    hipMemcpy(average_thread,dev_average_thread,N*sizeof(double),hipMemcpyDeviceToHost);


    for(int i=0; i<N; i++){
            cout<<average_thread[i]<<" "<<endl;
    }

	hipFree(dev_v_thread_seed);
	hipFree(dev_average_thread);

    delete[](v_thread_seed);
    delete(rng_Lcg);

}


