#include "hip/hip_runtime.h"
#include <header.hu>
__global__ void CombinedGenerator(thread_seed* dev_v_thread_seed, double* dev_average_thread){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    RNG* rng_Comb = new RNG_COMB(dev_v_thread_seed[tid].a, dev_v_thread_seed[tid].b, dev_v_thread_seed[tid].c, dev_v_thread_seed[tid].d);

    double sum = 0.;
    for(int i=0; i<N_PASSI; i++){
        sum += rng_Comb->get_uniform();
    }
    dev_average_thread[tid] = sum/N_PASSI;

    delete(rng_Comb);
}
